#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include <Eigen/Core>

#include <iostream>
#include <stdio.h>


static void HandleError( hipError_t err, const char *file, int line ){
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
    {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// CUDA Version
__global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N){
        out[idx] = v1[idx].dot(v2[idx]);
    }
    return;
}

__global__ void cu_calc_element_length(int num_element, int nodes_per_element, int *element_connectiviity, int num_node, float *nodes_x, float *nodes_y, float *nodes_z, float *element_length){
    // calulate the element length for each element
	float x_i, x_j, y_i, y_j, z_i, z_j;
	int i;
	for(i = 0; i<num_element; i++){
		x_i = nodes_x[element_connectiviity[i*nodes_per_element]]; 
		x_j = nodes_x[element_connectiviity[(i*nodes_per_element)+1]];
		y_i = nodes_y[element_connectiviity[i*nodes_per_element]];
		y_j = nodes_y[element_connectiviity[(i*nodes_per_element)+1]];
		z_i = nodes_z[element_connectiviity[i*nodes_per_element]];
		z_j = nodes_z[element_connectiviity[(i*nodes_per_element)+1]];
		element_length[i] = sqrtf(((x_j - x_i)*(x_j - x_i)) + ((y_j - y_i)*(y_j - y_i)) + ((z_j - z_i)*(z_j - z_i)));
	}
}

// The wrapper for the calling of the actual kernel
double dot(const std::vector<Eigen::Vector3d> & v1, const std::vector<Eigen::Vector3d> & v2){        
    int n = v1.size();
    double *ret = new double[n];

    // Allocate device arrays
    Eigen::Vector3d *dev_v1, *dev_v2;
    HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3d)*n));
    HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3d)*n));
    double* dev_ret;
    HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(double)*n));

    // Copy to device
    HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));

    // Dot product
    cu_dot<<<(n+1023)/1024, 1024>>>(dev_v1, dev_v2, dev_ret, n);
    
    // Copy to host
    HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(double)*n, hipMemcpyDeviceToHost));

    // Reduction of the array
    for (int i=1; i<n; ++i){
        ret[0] += ret[i];
    }

    // Return
    return ret[0];
}